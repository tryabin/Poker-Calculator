
#include <hip/hip_runtime.h>
extern "C" __device__ int get_suit(int card) {
    return card % 4;
}

extern "C" __device__ int get_rank(int card) {
    return (card - get_suit(card))/4;
}

extern "C" __device__ void sortCardsByRank(int cards[], int n) { 
    int i, key, j; 
    for (i = 1; i < n; i++) { 
        key = cards[i]; 
        j = i - 1; 
  
        /* Move elements of cards[0..i-1], that are 
          greater than key, to one position ahead 
          of their current position */
        while (j >= 0 && get_rank(cards[j]) > get_rank(key)) { 
            cards[j + 1] = cards[j]; 
            j = j - 1; 
        } 
        cards[j + 1] = key; 
    } 
} 