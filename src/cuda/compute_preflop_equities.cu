#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <phevaluator.h>

#define DECK_SIZE (52)
#define COMBO_DATA_SIZE (5)


extern "C" __global__ void compute_preflop_equities(int *holeCardCombos, int *outcomeTallies, int numberOfCombos,
                                                    short *binaries_by_id,
                                                    short *suitbit_by_id,
                                                    short *flush,
                                                    short *noflush7,
                                                    unsigned char *suits,
                                                    int *dp) {

    int posX = threadIdx.x + blockDim.x*blockIdx.x;
    int posY = threadIdx.y + blockDim.y*blockIdx.y;
    int threadNumber = posX + posY*blockDim.x*gridDim.x;
    int threadNumberInBlock = threadIdx.x + blockDim.x*threadIdx.y;

    if (threadNumber >= numberOfCombos) {
        return;
    }
 
    // int maxEvaluations = 8000;
    
    // Load data into the shared memory using the first thread in each block.
    extern __shared__ int sharedMemory[];
    int *dp_shared = (int*)&sharedMemory[0];
    unsigned char *suits_shared = (unsigned char*)&dp_shared[700];
    short *suitbit_by_id_shared = (short*)(&suits_shared[4609]+1); // shorts are aligned by increments of 2.
    short *binaries_by_id_shared = (short*)&suitbit_by_id_shared[52];
    
    if (threadNumberInBlock == 0) {
        // dp data
        for (int i = 0; i < 700; i++) {
            dp_shared[i] = dp[i];
        }
    
        // suits data
        for (int i = 0; i < 4609; i++) {
            suits_shared[i] = suits[i];
        }
        
        // suitbit_by_id data
        for (int i = 0; i < 52; i++) {
            suitbit_by_id_shared[i] = suitbit_by_id[i];
        }
        
        // binaries_by_id data
        for (int i = 0; i < 52; i++) {
            binaries_by_id_shared[i] = binaries_by_id[i];
        }       
    }
    __syncthreads();
        
    
    // Add the hole cards to the used cards array.
    int mainCard1 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 0];
    int mainCard2 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 1];
    int otherCard1 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 2];
    int otherCard2 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 3];

    // Initialize an array to store the used cards.
    bool usedCards[DECK_SIZE];
    for (int i = 0; i < DECK_SIZE; i++) {
        usedCards[i] = false;
    }
    usedCards[mainCard1] = true;
    usedCards[mainCard2] = true;
    usedCards[otherCard1] = true;
    usedCards[otherCard2] = true;
    
    
    // int numEvaluations = 0;
    
     // Card 1
    for (int card1 = 0; card1 < DECK_SIZE; card1++) { if (!usedCards[card1]) {

     // Card 2
    for (int card2 = card1+1; card2 < DECK_SIZE; card2++) { if (!usedCards[card2]) {

     // Card 3
    for (int card3 = card2+1; card3 < DECK_SIZE; card3++) { if (!usedCards[card3]) {

     // Card 4
    for (int card4 = card3+1; card4 < DECK_SIZE; card4++) { if (!usedCards[card4]) {

     // Card 5
    for (int card5 = card4+1; card5 < DECK_SIZE; card5++) { if (!usedCards[card5]) {
        
        int mainHandValue = evaluate_7cards(mainCard1, mainCard2, card1, card2, card3, card4, card5, 
                                            binaries_by_id_shared,
                                            suitbit_by_id_shared,
                                            flush,
                                            noflush7,
                                            suits_shared,
                                            dp_shared);
        int otherHandValue = evaluate_7cards(otherCard1, otherCard2, card1, card2, card3, card4, card5,
                                            binaries_by_id_shared,
                                            suitbit_by_id_shared,
                                            flush,
                                            noflush7,
                                            suits_shared,
                                            dp_shared);
                                            

        // Increment the win/loss/tie tallies.
        if (mainHandValue < otherHandValue) {
            outcomeTallies[threadNumber*3 + 0]++;
        }
        else if (mainHandValue > otherHandValue) {
            outcomeTallies[threadNumber*3 + 1]++;
        }
        else {
            outcomeTallies[threadNumber*3 + 2]++;
        }
        
        
        // numEvaluations++;
        // if (numEvaluations > maxEvaluations) {
            // return;
        // }

    }}}}}}}}}}
}

    



