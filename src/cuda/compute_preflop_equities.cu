#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <hip/hip_math_constants.h>
#include <phevaluator.h>
#include <util.h>
#include <get_straight_flush.h>

#define DECK_SIZE (52)
#define COMBO_DATA_SIZE (5)




extern "C" __global__ void compute_preflop_equities(int *holeCardCombos, int *outcomeTallies, int numberOfCombos,
                                                    short *binaries_by_id,
                                                    short *suitbit_by_id,
                                                    short *flush,
                                                    short *noflush7,
                                                    unsigned char *suits,
                                                    int *dp) {

    int posX = threadIdx.x + blockDim.x*blockIdx.x;
    int posY = threadIdx.y + blockDim.y*blockIdx.y;
    int threadNumber = posX + posY*blockDim.x*gridDim.x;
    int threadNumberInBlock = threadIdx.x + blockDim.x*threadIdx.y;

    if (threadNumber >= numberOfCombos) {
        return;
    }
 
    int maxEvaluations = 8000;
      
    // if (threadNumber == 0) {
        // int cards[] = {0, 4, 8, 12, 48, 5, 6};
        
        // for (int i = 0; i < 7; i++) {
            // printf("{%d %d} ", get_rank(cards[i]), get_suit(cards[i]));
        // }
        // printf("\n");
        
        // StraightFlushResult straightFlushResult = get_straight_flush_result(cards);
        
        // printf("straight flush found = %s\n", straightFlushResult.straightFlushExists ? "true" : "false");
        // if (straightFlushResult.straightFlushExists) {
            // printf("straight flush high card rank = %d\n", straightFlushResult.straightFlushHighCardRank);
        // }
    // }
    
    
    // Add the hole cards to the used cards array.
    int mainCard1 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 0];
    int mainCard2 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 1];
    int otherCard1 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 2];
    int otherCard2 = holeCardCombos[threadNumber*COMBO_DATA_SIZE + 3];

    // Initialize an array to store the used cards.
    bool usedCards[DECK_SIZE];
    for (int i = 0; i < DECK_SIZE; i++) {
        usedCards[i] = false;
    }
    usedCards[mainCard1] = true;
    usedCards[mainCard2] = true;
    usedCards[otherCard1] = true;
    usedCards[otherCard2] = true;
    
    
    int numEvaluations = 0;
    
     // Card 1
    for (int card1 = 0; card1 < DECK_SIZE; card1++) { if (!usedCards[card1]) {

     // Card 2
    for (int card2 = card1+1; card2 < DECK_SIZE; card2++) { if (!usedCards[card2]) {

     // Card 3
    for (int card3 = card2+1; card3 < DECK_SIZE; card3++) { if (!usedCards[card3]) {

     // Card 4
    for (int card4 = card3+1; card4 < DECK_SIZE; card4++) { if (!usedCards[card4]) {

     // Card 5
    for (int card5 = card4+1; card5 < DECK_SIZE; card5++) { if (!usedCards[card5]) {
        
        if (numEvaluations == maxEvaluations) {
            return;
        }
 
        int mainCards[] = {mainCard1, mainCard2, card1, card2, card3, card4, card5};
        int otherCards[] = {otherCard1, otherCard2, card1, card2, card3, card4, card5};
        
        // Check to see if there is a straight flush.
        StraightFlushResult straightFlushResultMain = get_straight_flush_result(mainCards);
        StraightFlushResult straightFlushResultOther = get_straight_flush_result(otherCards);
        
        // Win
        if (straightFlushResultMain.straightFlushExists && ! straightFlushResultOther.straightFlushExists) {
            outcomeTallies[threadNumber*3 + 0]++;
        }
        // Win
        else if (straightFlushResultMain.straightFlushExists && straightFlushResultOther.straightFlushExists && 
                 straightFlushResultMain.straightFlushHighCardRank > straightFlushResultOther.straightFlushHighCardRank) {
            outcomeTallies[threadNumber*3 + 0]++;
        }
        // Tie
        else if (!straightFlushResultMain.straightFlushExists && !straightFlushResultOther.straightFlushExists) {
            outcomeTallies[threadNumber*3 + 2]++;
        }
        // Tie
        else if (straightFlushResultMain.straightFlushExists && straightFlushResultOther.straightFlushExists &&
                 straightFlushResultMain.straightFlushHighCardRank == straightFlushResultOther.straightFlushHighCardRank) {
            outcomeTallies[threadNumber*3 + 2]++;
        }
        // Loss
        else {
            outcomeTallies[threadNumber*3 + 1]++;
        }

        
        numEvaluations++;


    }}}}}}}}}}
}

    



